#include "hip/hip_runtime.h"
#include <optix.h>
#include <math.h>
#include "rtx_params.h"


extern "C" static __constant__ Params params;

// min con closesthit
extern "C" __global__ void __raygen__rtx1() {
    //const uint3 idx = optixGetLaunchIndex();
    float &min = params.min;
    float &max = params.max;

    float3 ray_origin = make_float3(min, 0.0, 0.0);
    float3 ray_direction = make_float3(1.0, 0.0, 0.0);

    float tmin = 0;
    float tmax = max - min;
    float ray_time = 0;
    OptixVisibilityMask visibilityMask = 255;
    unsigned int rayFlags = OPTIX_RAY_FLAG_DISABLE_ANYHIT;
    unsigned int SBToffset = 0;
    unsigned int SBTstride = 0;
    unsigned int missSBTindex = 0;
    unsigned int payload = __float_as_uint(min);
    optixTrace(params.handle, ray_origin, ray_direction, tmin, tmax, ray_time,
            visibilityMask, rayFlags, SBToffset, SBTstride, missSBTindex, payload);

    *(params.output) = __uint_as_float(payload) + min;
}

extern "C" __global__ void  __closesthit__rtx() {
    float curr_tmax = optixGetRayTmax();
    optixSetPayload_0(__float_as_uint(curr_tmax));
}

extern "C" __global__ void  __miss__rtx() {
    optixSetPayload_0(__float_as_uint(INFINITY));
}




